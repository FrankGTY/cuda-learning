
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 512

void host_add(int *a, int *b, int *c) {
	for(int idx=0;idx<N;idx++)
		c[idx] = a[idx] + b[idx];
}

__global__ void device_add(int *a, int *b, int *c) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

//basically just fills the array with index.
void fill_array(int *data) {
	for(int idx=0;idx<N;idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int*c) {
	for(int idx=0;idx<N;idx++)
		printf("\n %d + %d  = %d",  a[idx] , b[idx], c[idx]);
}
int main(void) {
	int *a, *b, *c;
    int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); fill_array(a);
	b = (int *)malloc(size); fill_array(b);
	c = (int *)malloc(size);

	// Alloc space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    device_add<<<1,N>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    print_output(a,b,c);

	free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

	return 0;
}